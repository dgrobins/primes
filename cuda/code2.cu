#include <stdio.h>
#include <stdlib.h>

// cuda runtime
#include <hip/hip_runtime.h>

__global__ void kernel(int *a)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	a[idx] = idx;
	//var2: a[idx] = blockIdx.x;
	//var3: a[idx] = threadIdx.x;
}

int main()
{
	int dimx = 16;
	int num_bytes= dimx*sizeof(int);

	int *d_a=0, *h_a=0; // device and host pointers

	h_a = (int*) malloc(num_bytes);
	hipMalloc((void**) &d_a, num_bytes);

	if (h_a == 0 || d_a == 0)
	{
		printf("couldn't allocate memory\n");
		return 1;
	}

	hipMemset(d_a, 0, num_bytes);

	dim3 grid, block;
	block.x = 4;
	grid.x = dimx / block.x;

	kernel<<<grid, block>>>(d_a);

	hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);  // dest_ptr, src_ptr, direction (dev2host)

	for (int i=0; i < dimx; i++)
		printf("%d ", h_a[i]);
	printf("\n");

	free(h_a);
	hipFree(d_a);

	return 0;
}
